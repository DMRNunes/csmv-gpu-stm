#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <fstream>
#include <string>
#include <iostream>
#include <limits>
#include <string.h>
#include <unistd.h>
#include <cassert>
#include <inttypes.h>
#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <fcntl.h>
#include <unistd.h>
#include <cassert>
#include <inttypes.h>

#define SENDER_VERSION BASE_SEND

#include "../sync_lib/common.h"
#include "API.cuh"

/*
 * app specific config
 */
////////////////////
#define NUM_RECEIVER 1
#define MSG_SIZE_OFFLOAD 2
#define MSG_SIZE_MAX MSG_SIZE_OFFLOAD
///////////////////

#define ALG_ARG_DEF uint* token
#define ALG_ARG  token
#define PRINT_DEBUG_ 1

#define SERV_ARG_DEF TMmetadata* metadata, TXRecord* records, readSet* rs, writeSet* ws, warpResult* wRes, Statistics* stats, time_rate* times
#define SERV_ARG metadata, records, rs, ws, wRes, stats, times

#define PERF_METRICS 1

__device__ __forceinline__ void critcal_section(SERV_ARG_DEF, uint val0, int val1)
{

	uint tid = val0;
	uint timestamp = val1;
	int result;

	//validation
	//////////////////
#if PRINT_DEBUG_ == 0
	if(get_lane_id()==0) printf("\t\tS%d: recv %d %d\n", thread_id_x()/32, tid/32, timestamp);
#endif

	result=TXAddToRecord(metadata, records, rs, ws, stats, times, timestamp, tid);

#if PRINT_DEBUG_ == 0
	if(get_lane_id()==0) printf("\t\t\tS%d: sent %d %d\n", thread_id_x()/32, tid/32, result);
#endif
	
	wRes[tid/32].lane_result[tid%32]=result;
	__threadfence();
	if(get_lane_id() == 0)
		wRes[tid/32].valid_entry=1;

#if PRINT_DEBUG_ == 0
	if(get_lane_id()==0) printf("\t\t\t\tS%d: finished sending to %d\n", thread_id_x()/32, tid/32);
#endif

}

#include "../sync_lib/one_phase_def.h"
#include "../sync_lib/msg_config.h"
#include "../sync_lib/msg_aux.h"
#include "../sync_lib/msg_passing.h"
#include "../sync_lib/one_phase_server.h"

__global__ void client_kernel(gbc_t gbc, uint64_t seed, uint dataSize, VertionedDataItem* data, readSet* rs, writeSet* ws, warpResult* wRes, int* ro, int roSize, int upSize, 
								Statistics* stats, time_rate* times) {

	uint tid = threadIdx.x + blockIdx.x * blockDim.x;
	uint wid = tid/32;

	init_base(_offload_tail_ptr_cpy);

	__syncthreads();

	uint64_t state = seed+tid;
	int value, timestamp, addr, result;
	bool isAborted;
	int min, max;
	min = 10*tid;
	max = 10*(tid+1)-1;

	uint dst = 0;
	uint val0 = tid;
	int val1;

	uint valid_msg, retry=1;
	uint saved_write_ptr = NOT_FOUND;

	long long int start_time_writeback, stop_time_writeback;
	long long int start_time_commit, stop_time_commit;
	long long int start_time_tx, stop_time_tx;

	do
	{
#if PRINT_DEBUG == 0
		if(get_lane_id()==0) printf("C%d begin\n", tid/32);
#endif
		start_time_tx = clock64();
		timestamp=TXBegin(tid, ws, rs);
		isAborted=false;
		valid_msg = 1;

		//Read-Only TX
		if(ro[wid] == 1)
		{
			for(int i=0, value=0; i<roSize && isAborted==false; i++)
			{
				addr = RAND_R_FNC(state)%(max-min+1) + min;
				value+=TXRead(data, addr, timestamp, rs, ws, tid, &isAborted);
			}
			if(isAborted==true)
			{
				atomicAdd(&(stats->nbAbortsDataAge), 1);
				continue;
			}
			//printf("t%d: ro %d\n", id, value);
		}
		//Update TX
		else
		{
			for(int i=0; i<upSize && isAborted==false; i++)
			{
				addr = RAND_R_FNC(state)%(max-min+1) + min;
				value = TXRead(data, addr, timestamp, rs, ws, tid, &isAborted); 
				TXWrite(data, value-(tid*10+100), addr, ws, tid);

				addr = RAND_R_FNC(state)%(max-min+1) + min;
				value = TXRead(data, addr, timestamp, rs, ws, tid, &isAborted); 
				TXWrite(data, value+(tid*10+100), addr, ws, tid);
			}
			if(isAborted==true)
			{
				atomicAdd(&(stats->nbAbortsDataAge), 1);
				continue;
			}
		}
		
		/////////////////////////////
		//Commit process
		/////////////////////////////
		start_time_commit = clock64();
		if(ro[wid] == 1)
		{
			start_time_writeback = clock64();
			atomicAdd(&(stats->nbCommits), 1);
			stop_time_writeback = clock64();
			retry=0;
		}
		else
		{
			if(retry==0)
				val1=-1;
			else
				val1=timestamp;
			

	#if PRINT_DEBUG == 0
			if(get_lane_id()==0) printf("C%d: sending %d %d\n", tid/32, val0/32, val1);
	#endif

			if(get_lane_id()==0)
				saved_write_ptr = atomicAdd(&(gbc.write_ptr[dst]), 32);
			saved_write_ptr = shuffle_idx(saved_write_ptr, 0) + get_lane_id();
			do
			{
				if(base_send(gbc, dst, valid_msg, saved_write_ptr, _offload_tail_ptr_cpy, SEND_ARG))
					valid_msg = 0;
			}
			while(vote_ballot(valid_msg) != 0);

	//printf("C%d: sent\n", tid);	
			if(get_lane_id()==0)
				while(wRes[wid].valid_entry==0);
			result = wRes[wid].lane_result[get_lane_id()];

	#if PRINT_DEBUG == 0
			if(get_lane_id()==0) printf("C%d: recv %d\n", tid/32, result);
	#endif

			if(result != 0)
			{
				start_time_writeback = clock64();
				TXWriteBack(result, data, ws[tid]);
				stop_time_writeback = clock64();
				atomicAdd(&(stats->nbCommits), 1);
				retry=0;
			}
			//reset scoreboard
			if(get_lane_id()==0)
				wRes[wid].valid_entry=0;
		}
		stop_time_commit = clock64();

#if PRINT_DEBUG == 0	
		if(tid%32==0) printf("C%d: nbCommits: %d\n", tid/32, stats->nbCommits);
#endif

	}while(vote_ballot(retry) != 0);
	stop_time_tx = clock64();
	times[tid].commit = (float)(stop_time_commit - start_time_commit)/(stop_time_tx-start_time_tx);
	times[tid].dataWrite = (float)(stop_time_writeback - start_time_writeback)/(stop_time_tx-start_time_tx);

	//exit process
	base_exit(gbc);
}

__device__ void worker_thread(gbc_pack_t gbc_pack, SERV_ARG_DEF)
{
	uint m_warp_id = thread_id_x() / 32;

	VAR_BUF_DEF0
	uint stage_buf0 = 0;
	uint lock_id0;

	//for(stage_buf0=0; stage_buf0 < WORK_BUFF_SIZE_MAX; stage_buf0++)
	{		
		process_buffer_main_worker(gbc_pack, m_warp_id, 0, VAR_BUF0,
				stage_buf0,lock_id0, SERV_ARG);
	}
}


__global__ void server_kernel(gbc_pack_t gbc_pack, readSet* rs, writeSet* ws, TXRecord* records, warpResult* wRes, Statistics* stats, time_rate* times)
{
	__shared__ TMmetadata metadata;
	//__shared__ uint txNumber[TXRecordSize];

	init_recv(gbc_pack);
	gc_receiver_leader(gbc_pack);
	while(1)
	{
		worker_thread(gbc_pack, &metadata, records, rs, ws, wRes, stats, times);
	}
}


__global__ void parent_kernel(uint total_sender_bk, uint sender_block_size,	uint total_recevier_bk, uint recv_block_size, gbc_pack_t gbc_pack,
								uint64_t seed, uint dataSize, VertionedDataItem* data, readSet* rs, writeSet* ws, TXRecord* records, warpResult* wRes,
								int* ro, int roSize, int upSize, Statistics* stats, time_rate* times) {

//	for(int i=0; i<SCOREBOARD_SIZE/32; i++)
//		validSB[i]=0;
	hipStream_t s2;
	hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);

	server_kernel<<<total_recevier_bk, recv_block_size, 0, s2>>>(gbc_pack, rs, ws, records, wRes, stats, times);

	hipStream_t s1;
	hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);

	client_kernel<<<total_sender_bk, sender_block_size, 0, s1>>>(
			gbc_pack.gbc[CHANNEL_OFFLOAD], seed, dataSize, data, rs, ws, wRes,
			ro, roSize, upSize, stats, times);

}

void test_fine_grain_offloading(int seed, int dataSize, int client_block_size, int total_client_bk, int server_block_size, int roNum, int roSize, int upSize)
{

	int total_server_bk=1;
//	void (*server_kernel)(gbc_pack_t,
//	ALG_ARG_DEF);
//	server_kernel = server_one_phase;
	gbc_pack_t gbc_pack;
	create_gbc(gbc_pack, total_client_bk, client_block_size, server_block_size);
///////////////
	
	int* bankArray, *h_ro, *d_ro, aux;
	VertionedDataItem *h_data, *d_data;
	TXRecord* records;
	warpResult* wRes;
	readSet* rs;
	writeSet* ws;
	time_rate *h_times, *d_times;

	Statistics *h_stats, *d_stats;
	struct timespec t1,t2;
  	double elapsed_ms;

	h_ro = (int*) calloc(total_client_bk*client_block_size/32, sizeof(int));
	for(int i=0; i<roNum;)
	{
		aux = rand()%(total_client_bk*client_block_size/32);
		if(h_ro[aux]==0)
		{
			h_ro[aux]=1;
			i++;
		}
	}
	h_times = (time_rate*) calloc(total_client_bk*client_block_size,sizeof(time_rate));
	h_stats = (Statistics*)calloc(1,sizeof(Statistics));


	bankArray = (int*)malloc(dataSize*sizeof(int));
	for(int i=0; i<dataSize; i++)
	{
		bankArray[i]=1000;
	}
	//Allocate memory in the device
	hipError_t result;
	result = TXInit(bankArray, dataSize, client_block_size*total_client_bk, &h_data, &d_data, &rs, &ws, &records, &wRes);
	if(result != hipSuccess) fprintf(stderr, "Failed TM Initialization: %s\n", hipGetErrorString(result));
	result = hipMalloc((void **)&d_ro, total_client_bk*client_block_size/32*sizeof(int));
	if(result != hipSuccess) fprintf(stderr, "Failed to allocate d_ro: %s\n", hipGetErrorString(result));
	result = hipMalloc((void **)&d_stats, sizeof(Statistics));
	if(result != hipSuccess) fprintf(stderr, "Failed to allocate d_stats: %s\n", hipGetErrorString(result));
	result = hipMalloc((void **)&d_times, total_client_bk*client_block_size*sizeof(time_rate));
	if(result != hipSuccess) fprintf(stderr, "Failed to allocate d_ratio: %s\n", hipGetErrorString(result));

	//transfer the necessary data from the host to the device
	hipMemcpy(d_ro, h_ro, total_client_bk*client_block_size/32*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_times, h_times, total_client_bk*client_block_size*sizeof(time_rate), hipMemcpyHostToDevice);
	hipMemcpy(d_stats, h_stats, sizeof(Statistics), hipMemcpyHostToDevice);

	///////////////
	//kernel stuff
	clock_gettime(CLOCK_REALTIME, &t1);
	{
		//hipDeviceSynchronize();

		parent_kernel<<<1, 1>>>(total_client_bk, client_block_size,
		total_server_bk, server_block_size, gbc_pack,
		1, dataSize, d_data, rs, ws, records, wRes,
		d_ro, roSize, upSize, d_stats, d_times);
		hipDeviceSynchronize();
	}
	clock_gettime(CLOCK_REALTIME, &t2);
	//////////////  


	free_gbc(gbc_pack);
	TXEnd(dataSize, h_data, &d_data, &rs, &ws, &wRes);

	//Copy metric data back to the host
	hipMemcpy(h_stats, d_stats, sizeof(Statistics), hipMemcpyDeviceToHost);
  	hipMemcpy(h_times, d_times, total_client_bk*client_block_size*sizeof(time_rate), hipMemcpyDeviceToHost);

  	float avg_commit=0.0, avg_wb=0.0, avg_val=0.0, avg_rwb=0.0;
	for(int i=0; i<total_client_bk*client_block_size; i++)
	{
		avg_commit 	+= h_times[i].commit;
		avg_wb 		+= h_times[i].dataWrite;
		avg_val		+= h_times[i].validation;
		avg_rwb		+= h_times[i].recordWrite;
	}
	avg_commit 	/= (total_client_bk*client_block_size);
	avg_wb 		/= (total_client_bk*client_block_size);
	avg_val 	/= (total_client_bk*client_block_size);
	avg_rwb 	/= (total_client_bk*client_block_size);


	elapsed_ms = (t2.tv_sec*1000.0 + t2.tv_nsec/1000000.0) - (t1.tv_sec*1000.0 + t1.tv_nsec/1000000.0);
  	//printf("Elapsed time: %.3f [ms]\n",elapsed_ms);
	int nbAborts = h_stats->nbAbortsDataAge + h_stats->nbAbortsRecordAge + h_stats->nbAbortsReadWrite + h_stats->nbAbortsWriteWrite;

	printf("%.3f\t%.3f\t%.3f\t%.3f\t%.3f\t%.3f\t%.3f\t%.3f\t%.3f\t%.3f\n", (float)nbAborts/(nbAborts+h_stats->nbCommits), h_stats->nbCommits/elapsed_ms*1000.0, 
												(float)h_stats->nbAbortsDataAge/(nbAborts+h_stats->nbCommits), (float)h_stats->nbAbortsRecordAge/(nbAborts+h_stats->nbCommits), 
												(float)h_stats->nbAbortsReadWrite/(nbAborts+h_stats->nbCommits), (float)h_stats->nbAbortsWriteWrite/(nbAborts+h_stats->nbCommits),
												avg_commit, avg_wb, avg_val, avg_rwb);
	free(h_ro);
	free(h_stats);
	free(h_times);
	hipFree(d_ro);
	hipFree(d_stats);
	hipFree(d_times);
}

int main(int argc, char *argv[]) {

	int client_block_size, server_block_size;
	int total_client_bk;
	int dataSize, roNum, roSize, upSize;

	if(argc != 8)
	{
		printf("Usage: <dataSize> <client_block_size> <total_client_blocks> <server_block_size> <RO%> <ROlen> <UPlen>\n");
		return 1;
	}

	dataSize			= atoi(argv[1]);
	client_block_size	= atoi(argv[2]);
	total_client_bk 	= atoi(argv[3]);
	server_block_size	= atoi(argv[4]);
	roNum 				= (atoi(argv[5])/100.0) * total_client_bk*client_block_size/32;
	roSize 				= atoi(argv[6]);
	upSize				= atoi(argv[7]);

	dataSize=10*total_client_bk*client_block_size;
	if(roNum>total_client_bk*client_block_size/32) roNum=total_client_bk*client_block_size/32;

	hipError_t result;
	result = hipSetDevice(0);
	if(result != hipSuccess) fprintf(stderr, "Failed to set Device: %s\n", hipGetErrorString(result));
	
	for (int i = 0; i < 1; i++) {
		test_fine_grain_offloading(i, dataSize, client_block_size, total_client_bk, server_block_size, roNum, roSize, upSize);
	}
	return 0;
}
